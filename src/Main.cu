#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "Init.cuh"
#include "OneSweep.cuh"

const uint32_t size = (1 << 28);
const uint32_t testIterations = 25;

//Disable this when increasing test iterations, otherwise will be too slow
//because of the device to host readback speed
const uint32_t performValidation = true;

const uint32_t radix = 256;
const uint32_t radixPasses = 4;
const uint32_t partitionSize = 7680;
const uint32_t globalHistPartitionSize = 65536;
const uint32_t globalHistThreads = 128;
const uint32_t binningThreads = 512;			//2080 super seems to really like 512 
const uint32_t binningThreadblocks = (size + partitionSize - 1) / partitionSize;
const uint32_t globalHistThreadblocks = (size + globalHistPartitionSize - 1) / globalHistPartitionSize;

uint32_t* sort;
uint32_t* alt;
uint32_t* index;
uint32_t* globalHistogram;
uint32_t* firstPassHistogram;
uint32_t* secPassHistogram;
uint32_t* thirdPassHistogram;
uint32_t* fourthPassHistogram;

void InitMemory()
{
	hipMemset(index, 0, radixPasses * sizeof(uint32_t));
	hipMemset(globalHistogram, 0, radix * radixPasses * sizeof(uint32_t));
	hipMemset(firstPassHistogram, 0, radix * binningThreadblocks * sizeof(uint32_t));
	hipMemset(secPassHistogram, 0, radix * binningThreadblocks * sizeof(uint32_t));
	hipMemset(thirdPassHistogram, 0, radix * binningThreadblocks * sizeof(uint32_t));
	hipMemset(fourthPassHistogram, 0, radix * binningThreadblocks * sizeof(uint32_t));
}

void DispatchKernels()
{
	InitMemory();

	hipDeviceSynchronize();

	GlobalHistogram <<<globalHistThreadblocks, globalHistThreads >>> (sort, globalHistogram, size);

	Scan <<<radixPasses, radix >>> (globalHistogram, firstPassHistogram, secPassHistogram,
		thirdPassHistogram, fourthPassHistogram);

	DigitBinningPass <<<binningThreadblocks, binningThreads >>> (sort, alt, firstPassHistogram,
		index, size, 0);

	DigitBinningPass <<<binningThreadblocks, binningThreads >>> (alt, sort, secPassHistogram,
		index, size, 8);

	DigitBinningPass <<<binningThreadblocks, binningThreads >>> (sort, alt, thirdPassHistogram,
		index, size, 16);

	DigitBinningPass <<<binningThreadblocks, binningThreads >>> (alt, sort, fourthPassHistogram,
		index, size, 24);
}

//Test for correctness
void ValidationTest()
{
	printf("Beginning VALIDATION tests at size %u and %u iterations. \n", size, testIterations);
	uint32_t* validationArray = new uint32_t[size];
	int testsPassed = 0;

	for (uint32_t i = 1; i <= testIterations; ++i)
	{
		InitRandom <<<256, 1024>>> (sort, size, i);
		DispatchKernels();
		hipDeviceSynchronize();
		hipMemcpy(validationArray, sort, size * sizeof(uint32_t), hipMemcpyDeviceToHost);

		bool isCorrect = true;
		for (uint32_t k = 1; k < size; ++k)
		{
			if (validationArray[k] < validationArray[k - 1])
			{
				isCorrect = false;
				break;
			}
		}

		if (isCorrect)
			testsPassed++;
		else
			printf("Test iteration %d failed.", i);
	}

	printf("%d/%d tests passed.\n", testsPassed, testIterations);
	delete[] validationArray;
}

//Discard the first result to prep caches and TLB
void TimingTest()
{
	printf("Beginning TIMING tests at size %u and %u iterations. \n", size, testIterations);
	printf("Running ");

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float totalTime = 0.0f;
	for (uint32_t i = 0; i <= testIterations; ++i)
	{
		InitRandom <<<256, 1024>>> (sort, size, i);
		hipDeviceSynchronize();
		hipEventRecord(start);
		DispatchKernels();
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float millis;
		hipEventElapsedTime(&millis, start, stop);
		if (i)
			totalTime += millis;

		if ((i & 15) == 0)
			printf(". ");
	}

	printf("\n");
	totalTime /= 1000.0f;
	printf("Total time elapsed: %f\n", totalTime);
	printf("Estimated speed at %u 32-bit elements: %E keys/sec\n", size, size / totalTime * testIterations);
}

int main()
{
	hipMalloc(&sort, size * sizeof(uint32_t));
	hipMalloc(&alt, size * sizeof(uint32_t));
	hipMalloc(&index, radixPasses * sizeof(uint32_t));
	hipMalloc(&globalHistogram, radix * radixPasses * sizeof(uint32_t));
	hipMalloc(&firstPassHistogram, binningThreadblocks * radix * sizeof(uint32_t));
	hipMalloc(&secPassHistogram, binningThreadblocks * radix * sizeof(uint32_t));
	hipMalloc(&thirdPassHistogram, binningThreadblocks * radix * sizeof(uint32_t));
	hipMalloc(&fourthPassHistogram, binningThreadblocks * radix * sizeof(uint32_t));

	if (performValidation)
		ValidationTest();
	TimingTest();

	hipFree(sort);
	hipFree(alt);
	hipFree(index);
	hipFree(globalHistogram);
	hipFree(firstPassHistogram);
	hipFree(secPassHistogram);
	hipFree(thirdPassHistogram);
	hipFree(fourthPassHistogram);
}